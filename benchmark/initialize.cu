#include "hip/hip_runtime.h"
// Copyright (C) 2022 Intel Corporation
// SPDX-License-Identifier: BSD-3-Clause

#include "initialize.hpp"
#define CUDA
#include "signal.hpp"
#undef CUDA
#include <algorithm>
#include <complex>
#include <thrust/complex.h>

template <typename T> struct real_type {
    using type = T;
};
template <typename T> struct real_type<thrust::complex<T>> {
    using type = T;
};

template <typename U, typename V, bool Inverse>
__global__ void init_kernel(V *x, unsigned int M, unsigned int N, unsigned int K, unsigned int s0,
                            unsigned int s1, unsigned int s2) {
    using real_type = typename real_type<U>::type;
    auto k = threadIdx.x + blockIdx.x * blockDim.x;
    auto n = blockIdx.y;
    auto m = blockIdx.z;
    if (k < K) {
        auto s = signal<U>(N, m + k, real_type(1.0) + k / real_type(K));
        if constexpr (Inverse) {
            x[k * s0 + n * s1 + m * s2] = s.X(n);
        } else {
            x[k * s0 + n * s1 + m * s2] = s.x(n) / real_type(N);
        }
    }
}

template <typename T> struct cu_type {
    using type = T;
};
template <typename T> struct cu_type<std::complex<T>> {
    using type = thrust::complex<T>;
};

template <typename U, typename V> void initialize(tensor<U, 3u> x, tensor<V, 3u> X, bool inverse) {
    unsigned int N = x.shape(1);
    auto shape = inverse ? X.shape() : x.shape();
    unsigned int K = shape[0];
    unsigned int NN = shape[1];
    unsigned int M = shape[2];
    unsigned int Kb = std::min(128u, K);
    auto num_threads = dim3(Kb, 1, 1);
    auto num_blocks = dim3((K - 1) / Kb + 1, NN, M);
    if (inverse) {
        init_kernel<typename cu_type<U>::type, typename cu_type<V>::type, true>
            <<<num_blocks, num_threads>>>(reinterpret_cast<typename cu_type<V>::type *>(X.data()),
                                          M, N, K, X.stride(0), X.stride(1), X.stride(2));
    } else {
        init_kernel<typename cu_type<U>::type, typename cu_type<U>::type, false>
            <<<num_blocks, num_threads>>>(reinterpret_cast<typename cu_type<U>::type *>(x.data()),
                                          M, N, K, x.stride(0), x.stride(1), x.stride(2));
    }
    hipDeviceSynchronize();
}

template void initialize<std::complex<float>, std::complex<float>>(tensor<std::complex<float>, 3u>,
                                                                   tensor<std::complex<float>, 3u>,
                                                                   bool);
template void initialize<float, std::complex<float>>(tensor<float, 3u>,
                                                     tensor<std::complex<float>, 3u>, bool);
template void
initialize<std::complex<double>, std::complex<double>>(tensor<std::complex<double>, 3u>,
                                                       tensor<std::complex<double>, 3u>, bool);
template void initialize<double, std::complex<double>>(tensor<double, 3u>,
                                                       tensor<std::complex<double>, 3u>, bool);

